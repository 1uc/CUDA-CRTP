#include "hip/hip_runtime.h"
/* Just some trivial tests.
 *
 * Authors: Luc Grosheintz <forbugrep@zoho.com>
 *    Date: 2016-04-18
 */
#include <stdio.h>
#include <iostream>
#include <sstream>

void checkCudaError( char const * const file, const int line );
void cudaFinalize(void);

#define SANE (checkCudaError(__FILE__, __LINE__))

template<class E>
class CRTPBase {
  public:
#ifdef HAS_USER_DEFINED_CAST
    __host__ __device__ __inline__
    operator E const&() const {
      return static_cast<const E &>(*this);
    }
#endif
};

class Impl : public CRTPBase<Impl> {
  public:
    double x;

  public:
    __host__ __device__ __inline__
    Impl(double x)
      : x(x)
    { }

    template<class E>
    __host__ __device__ __inline__
    void operator=(const CRTPBase<E> &e_) {
      const E& e = static_cast<const E&>(e_);
      x = e.x;
    }
};


__global__
void crtp_on_device_kernel(double * ret) {
  // The next three line will be referred to as (1)
  Impl x(1.0);
  CRTPBase<Impl> &e = x;
  x = e;

  *ret = x.x;
}

void crtp_on_host() {
  // Note, these three lines are one-to-one copy of (1)
  Impl x(1.0);
  CRTPBase<Impl> &e = x;
  x = e;

  printf("HURRAY, for the host.\n");
}

void crtp_on_device() {
  double * foo = NULL;
  hipMalloc(&foo, sizeof(double));                                        SANE;

  crtp_on_device_kernel<<<1, 1>>>(foo);                                    SANE;
  printf("HURRAY, for the device.\n");

  hipFree(foo);                                                           SANE;
  cudaFinalize();
}


int main() {
  crtp_on_host();
  crtp_on_device();

  return 0;
}

/// Clean up CUDA
void cudaFinalize(void){
  checkCudaError("before exit", -1);

  std::cout << ".. No CUDA-errors detected.\n";
  hipDeviceReset();
}

/// Checks whether a CUDA error has been raised.
void checkCudaError(char const * const file, const int line ){
  hipDeviceSynchronize();
  if( hipPeekAtLastError() != hipSuccess ){
    std::stringstream ss;
    ss << "!! Error: " << file << ": " << line << ": "
       << hipGetErrorString(hipPeekAtLastError())
       << " error no.: " << hipPeekAtLastError();
    ss << "\n";

    std::cout << ss.str();
    exit(EXIT_FAILURE);
  }
}
